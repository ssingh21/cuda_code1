#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <fstream>
#include <string>
#include <cstdlib>
#include <math.h>
#include <algorithm>
#include <bitset>
#include <iomanip>
 
using namespace std;

__global__ void twodimconvol(float *a, float *h, float *c, int kY, int kX, int dY, int dX,int newx,int newy)
{	
	//Launching X and Y threadIDS for computation
	int x=blockIdx.x*blockDim.x + threadIdx.x;
	int y=blockIdx.y*blockDim.y + threadIdx.y;

	int i,j;
	//Sum variable used for final result convolution at (i,j)
	float sum = 0.0;                            
	//IF Statement used because I have by default 16 by 16 blksize incase op is small thenfor that case it is used
	if(x < newx){
	   if(y < newy){
           for(i=0;i < kY; ++i)      
           {
                for(j=0; j < kX; ++j)  
                {
			
                        sum += h[kX * i + j]*a[((y-i+kY-1)*(dX))+(x-j + kX-1)];
                }
            }
            c[(dX-(kX-1)) * y + x] = sum;
	}
      }
        
}


int main (int argc, char* argv[])
{
	int dY, dX,kY, kX;
	ifstream infile;
  	int stat1 = 0,stat2 = 0,col1 = 0,col2 = 0;
 	 infile.open(argv[1]); // open a file
	ifstream file2;
	file2.open(argv[1]);
	 string A;
   while (!file2.eof())
  {
   char buffer[512];
   file2.getline(buffer,512);
    if(strcmp(buffer,"") == 0){
	break;
    }	
    else{
	int n = 0;
    		// array to store memory addresses of the tokens in buf
    		const char* token[250] = {}; // initialize to 0
    
   		 // parse the line
   		 token[0] = strtok(buffer," "); // first token
   		 if (token[0]) // zero if line is blank
   		 {
    		  for (n = 1; n < 250; n++)
    		  {
    	 	   token[n] = strtok(0, " "); // subsequent tokens
    	    	   if (!token[n]) break; // no more tokens
    	  	  }
    	 	 col2 = n;
    		}
    	// process (print) the tokens
    		for (int i = 0; i < n; i++){ // n = #of tokens
	
      			//cout << "Token[" << i << "] = " << token[i] << endl;
       			stat2++;
    		}
		//cout << endl;
	}
    }
   while (!file2.eof())
  {
   char buffer[512];
   file2.getline(buffer,512);
		int n = 0;
    		// array to store memory addresses of the tokens in buf
    		const char* token[250] = {}; // initialize to 0
    
   		 // parse the line
   		 token[0] = strtok(buffer," "); // first token
   		 if (token[0]) // zero if line is blank
   		 {
    		  for (n = 1; n < 250; n++)
    		  {
    	 	   token[n] = strtok(0, " "); // subsequent tokens
    	    	   if (!token[n]) break; // no more tokens
    	  	  }
    	 	 col1 = n;
    		}
    	// process (print) the tokens
    		for (int i = 0; i < n; i++){ // n = #of tokens
	
      			//cout << "Token[" << i << "] = " << token[i] << endl;
       			//check1[stat1] = atof(token[i]);
			//cout << check1[stat1] << endl;
       			stat1++;
    		}
		//cout << endl;
		if(file2.eof())
			break;
    }
    //cout << stat1 << " " << stat2 << " " << col1 << " " << col2 << endl;
    //Allocating matrix a1 and a2 based on file parsed above which gives us dimension
    float *a1,*a2;
    a1 = new float [stat2];
    a2 = new float [stat1];
    for(int i = 0;i < stat2 ;i++)
	a1[i] = 0.0;
    for(int i = 0;i < stat1;i++)
	a2[i] = 0.0;
    int dum1=0,dum2=0;

  
   //Actual read of matrices a1 and a2
   while (!infile.eof())
  {
   char buffer[512];
   infile.getline(buffer,512);
    if(strcmp(buffer,"") == 0){
	break;
    }	
    else{
	int n = 0;
    		// array to store memory addresses of the tokens in buf
    		const char* token[250] = {}; // initialize to 0
    
   		 // parse the line
   		 token[0] = strtok(buffer," "); // first token
   		 if (token[0]) // zero if line is blank
   		 {
    		  for (n = 1; n < 250; n++)
    		  {
    	 	   token[n] = strtok(0, " "); // subsequent tokens
    	    	   if (!token[n]) break; // no more tokens
    	  	  }
    	 	 //col2 = n;
    		}
    	// process (print) the tokens
    		for (int i = 0; i < n; i++){ // n = #of tokens
	
      			//cout << "Token[" << i << "] = " << token[i] << endl;
       			a1[dum1] = atof(token[i]);
			//cout << check2[stat2] << endl;
       			dum1++;
    		}
		//cout << endl;
	}
    }
   while (!infile.eof())
  {
   char buffer[512];
   infile.getline(buffer,512);
		int n = 0;
    		// array to store memory addresses of the tokens in buf
    		const char* token[250] = {}; // initialize to 0
    
   		 // parse the line
   		 token[0] = strtok(buffer," "); // first token
   		 if (token[0]) // zero if line is blank
   		 {
    		  for (n = 1; n < 250; n++)
    		  {
    	 	   token[n] = strtok(0, " "); // subsequent tokens
    	    	   if (!token[n]) break; // no more tokens
    	  	  }
    	 	 //col1 = n;
    		}
    	// process (print) the tokens
    		for (int i = 0; i < n; i++){ // n = #of tokens
	
      			//cout << "Token[" << i << "] = " << token[i] << endl;
       			a2[dum2] = atof(token[i]);
			//cout << check1[stat1] << endl;
       			dum2++;
    		}
		//cout << endl;
		if(infile.eof())
			break;
    }
 	//cout << stat1 << "  " << col1 << " " << stat2 << " " << col2 << endl;	
	int newX,newY;
	for(int i = 0;i < stat2 ;i++){
		//cout << a1[i] << endl;
	}
	int ro1 = stat1/col1; int ro2 = stat2/col2;
	newX = col1+col2-1;
	newY = ro1+ro2-1;
	dX=newX+col1-1;
	dY=newY+ro1-1;
 	kY=ro1;
	kX=col1;
	
	//Padding input matrix such that the convolution formulae is valid for all values
	

	//Host Input Vector
	float *host_a;
	//h_a = new float [dataSizeX*dataSizeY];
	float *host_h;
	//Host Output Vector
	float *host_c;

	//Device Input Vector
	float *device_a;
	float *device_h;
	//Host output Vector
	float *device_c;

	//Memory allocation for Host
	host_a = (float*)malloc((dX*dY)*sizeof(float));
	host_h = (float*)malloc((kY*kX)*sizeof(float));
	host_c = (float*)malloc((newX*newY)*sizeof(float));

	//Memory allocation for device
	hipMalloc(&device_a, (dX*dY)*sizeof(float));
	hipMalloc(&device_h, (kY*kX)*sizeof(float));
	hipMalloc(&device_c, (newX*newY)*sizeof(float));
        
	for(int i=0;i<dY;i++){
		for(int j=0;j<dX;j++){
			host_a[i*dX + j] = 0.0;
		}
	} 
        
	//Padding happens here on input matrix 2*(kY-1) rows added and 2*(kX-1) collumns added			
	//Intialize on Host
	int set1 = kX-1;
	int set2 = kY-1;
	int countset = 0;
	for(int i = set2;i <(dY-(kY-1)) ;i++){
		for(int j=0;j < set1;j++){
		   	host_a[i*dX + j] = 0.0;
		}
		for(int j = set1;j<(set1+col2);j++){
			host_a[i*dX + j] = a1[countset];
			countset++;	
		}
	}
					
         

	for(int i=0;i<(kY*kX);i++){			
		host_h[i] = a2[i];	
	}
	
		

	//Transfer to device
	hipMemcpy( device_a, host_a, (dX*dY)*sizeof(float), hipMemcpyHostToDevice);
    	hipMemcpy( device_h, host_h, (kY*kX)*sizeof(float), hipMemcpyHostToDevice);
	float x = newX;
       float y = newY;
       //Calculating GridSize required for computations
       int sizingX = (int)ceil((float)((float)(x)/16));
    int sizingY = (int)ceil((float)((float)(y)/16));
   
      dim3 blockSize(16,16);
     dim3 gridSize(sizingX,sizingY);
    twodimconvol<<<gridSize, blockSize>>>(device_a, device_h, device_c, kY, kX,dY, dX,newX,newY);

    // Copy array back to host
    hipMemcpy( host_c, device_c, (newX*newY)*sizeof(float), hipMemcpyDeviceToHost );

	// Release device memory
	hipFree(device_a);
	hipFree(device_h);
	hipFree(device_c);
	int i,j;
	for(int j=0;j<newY;j++){
		for(int i=0;i<newX;i++){			
			cout << fixed << setprecision(1) << host_c[j*newX + i] << " ";
		}
		cout << endl;
	}

 
	// Release host memory
        free(host_a);
        free(host_h);
        free(host_c);
 
        return 0;
	

}
